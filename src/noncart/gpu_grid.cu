#include "hip/hip_runtime.h"
/* Copyright 2022. Uecker Lab. University Medical Center Göttingen.
 * All rights reserved. Use of this source code is governed by
 * a BSD-style license which can be found in the LICENSE file.
 *
 * Authors: Moritz Blumenthal
 */

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <assert.h>

#include "misc/misc.h"

#include "num/multind.h"
#include "num/gpu_misc.h"
#include "num/gpuops.h"

#include "noncart/grid.h"
#include "gpu_grid.h"

__device__ hipFloatComplex zexp(hipFloatComplex x)
{
	float sc = expf(hipCrealf(x));
	float si;
	float co;
	sincosf(hipCimagf(x), &si, &co);
	return make_hipFloatComplex(sc * co, sc * si);
}

struct linphase_conf {

	long dims[3];
	long tot;
	float shifts[3];
	long N;
	float cn;
	float scale;
	_Bool conj;
	_Bool fmac;
};

__global__ void kern_apply_linphases_3D(struct linphase_conf c, hipFloatComplex* dst, const hipFloatComplex* src)
{
	int startX = threadIdx.x + blockDim.x * blockIdx.x;
	int strideX = blockDim.x * gridDim.x;

	int startY = threadIdx.y + blockDim.y * blockIdx.y;
	int strideY = blockDim.y * gridDim.y;

	int startZ = threadIdx.z + blockDim.z * blockIdx.z;
	int strideZ = blockDim.z * gridDim.z;

	for (long z = startZ; z < c.dims[2]; z += strideZ)
		for (long y = startY; y < c.dims[1]; y += strideY)
			for (long x = startX; x < c.dims[0]; x +=strideX) {

				long pos[3] = { x, y, z };
				long idx = x + c.dims[0] * (y + c.dims[1] * z);
				
				float val = c.cn;

				for (int n = 0; n < 3; n++)
					val += pos[n] * c.shifts[n];

				if (c.conj)
					val = -val;
				
				hipFloatComplex cval = make_hipFloatComplex(0, val);
				cval = zexp(cval);

				cval.x *= c.scale;
				cval.y *= c.scale;

				if (c.fmac) {

					for (long i = 0; i < c.N; i++)
						dst[idx + i * c.tot] = hipCaddf(dst[idx + i * c.tot], hipCmulf(src[idx + i * c.tot], cval));
				} else {

					for (long i = 0; i < c.N; i++)
						dst[idx + i * c.tot] = hipCmulf(src[idx + i * c.tot], cval);
				}
			}
}



extern "C" void cuda_apply_linphases_3D(int N, const long img_dims[], const float shifts[3], _Complex float* dst, const _Complex float* src, _Bool conj, _Bool fmac, float scale)
{
	struct linphase_conf c;

	c.cn = 0;
	c.tot = 1;
	c.N = 1;
	c.scale = scale;
	c.conj = conj;
	c.fmac = fmac;

	for (int n = 0; n < 3; n++) {

		c.shifts[n] = 2. * M_PI * (float)(shifts[n]) / ((float)img_dims[n]);
		c.cn -= c.shifts[n] * (float)img_dims[n] / 2.;
		
		c.dims[n] = img_dims[n];
		c.tot *= c.dims[n];
	}

	c.N = md_calc_size(N - 3, img_dims + 3);

	const void* func = (const void*)kern_apply_linphases_3D;
	kern_apply_linphases_3D<<<getGridSize3(c.dims, func), getBlockSize3(c.dims, (const void*)func), 0, cuda_get_stream()>>>(c, (hipFloatComplex*)dst, (const hipFloatComplex*)src);
}

